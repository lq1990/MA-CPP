#include "hip/hip_runtime.h"
﻿#include "Cuda4RNN.h"

void trainMultiThread(
	float* lossAllVec,
	Para* para
)
{
	int num_sces = para->num_sces[0];
	int total_epoches = para->total_epoches[0];
	int n_features = para->n_features[0];
	int n_hidden = para->n_hidden[0];
	int n_output_classes = para->n_output_classes[0];
	float alpha = para->alpha[0];
	float score_min = para->score_min[0];
	float score_max = para->score_max[0];
	{
		cout << "train begins" << endl << "alpha: " << alpha << endl;
		cout << "total_epoches: " << total_epoches << endl 
			<< "n_features: " << n_features << endl;
		cout << "n_hidden: " << n_hidden << endl 
			<< "n_output_classes: " << n_output_classes << endl;
		cout << "score_min: " << score_min << endl 
			<< "score_max: " << score_max << endl;
	}

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// 先用 标准SGD优化，使用1个cpu线程
	float* loss_one_epoch;
	float* loss_mean_each_epoch;
	float* true_false;
	float* accuracy_each_epoch;
	float loss = 0.f;
	hipMalloc((void**)&loss_one_epoch, total_epoches * sizeof(float));
	hipMalloc((void**)&loss_mean_each_epoch, total_epoches * sizeof(float));
	hipMalloc((void**)&true_false, total_epoches * sizeof(float));
	hipMalloc((void**)&accuracy_each_epoch, total_epoches * sizeof(float));

	int Nmax = para->d_Nmax[0];
	int M = para->sces_data_mn[0];
	float* sce_item_data;
	hipMalloc((void**)&sce_item_data, M*Nmax*sizeof(float));
	float* hprev;
	hipMalloc((void**)&hprev, n_hidden * sizeof(float));

	for (int i = 0; i < total_epoches/100; i++)
	{
		/**
			loop over each scenario
		*/
		for(int item = 0; item < num_sces/5; item++)
		{
			// ---------- 先取出一个 场景数据 训练 RNN ------------
			float id0 = para->sces_id_score[item*2 + 0]; 
			float score0 = para->sces_id_score[item*2 + 1];
			int sce0_M = para->sces_data_mn[item * 2 + 0]; 
			int sce0_N = para->sces_data_mn[item * 2 + 1];

			int beginIdx = para->sces_data_idx_begin[item];
			int endIdx = para->sces_data_idx_begin[item + 1];
			gpu_copy(sce_item_data, 0, para->sces_data, beginIdx, endIdx);

			//gpu_clear_arr(true_false, total_epoches);//true_false.clear();
			//gpu_clear_arr(loss_one_epoch, total_epoches);//loss_one_epoch.clear();

			// set dP 0
			gpu_fill(para->dWxh, n_hidden * n_features, 0.f);
			gpu_fill(para->dWhh, n_hidden * n_hidden, 0.f);
			gpu_fill(para->dWhy, n_hidden * n_output_classes, 0.f);
			gpu_fill(para->dbh,  n_hidden, 0.f);
			gpu_fill(para->dby,  n_output_classes, 0.f);

			gpu_fill(hprev, n_hidden, 0.f);
		
			lossFun(handle, 
				sce_item_data, sce0_M, sce0_N,
				score0, 
				hprev, true_false, 
				loss,
				para);

			cout << "epoch: " << i << ", #sce: "<< item << ", loss: " << loss << endl;

			sgd(handle, para);
		}

		/*if (i % 5 == 0)
		{
			
		}*/
	}

	// lossVec mean, accu

	// free resource
	;{
		deInitPara(para);
		hipFree(hprev); 
		hipFree(sce_item_data);
		hipFree(loss_one_epoch);
		hipFree(loss_mean_each_epoch);
		hipFree(true_false);
		hipFree(accuracy_each_epoch);
		cout << "free over in train fn \n";
	}
	hipblasDestroy(handle);
}


/*
	arma::mat in RNN-v2 => device_vector.

	inputs: data of a scenario
	M: n_rows of orig. inputs. 目前的 M=17 即signals的数目
	N: n_cols of orig. inputs. N 是matlab中matDataZScore的行数即time步

	注意：参数中有struct，当调用这个fn时，应先 hipMallocManaged struct
*/
void lossFun(
	hipblasHandle_t handle,
	float* inputs, int M, int N,
	float score,
	float* hprev,
	float* true_false,
	float& loss,
	Para* para
)
{
	int total_epoches = para->total_epoches[0];
	int n_features = para->n_features[0];
	int n_hidden = para->n_hidden[0];
	int n_output_classes = para->n_output_classes[0];
	float alpha = para->alpha[0];
	float score_min = para->score_min[0];
	float score_max = para->score_max[0];
	int Nmax = para->Nmax[0]; // 所有场景的N中最大的数值

	int idx1_targets = -1;
	float* targets = score2onehot(score, 
		idx1_targets, n_output_classes, score_min, score_max);

	// hs[-1] = hprev;
	gpu_set_col(para->hs, n_hidden, Nmax, -1+1, hprev);
	loss = 0.f;

	// ---------------- forward pass -------------
	for (int t = 0; t < N; t++)
	{
		// ----- xs[t] = inputs.row(t).t(); -----
		gpu_get_col(inputs, M, N, t, para->tmp_d_vec);  // tmp saves xs[t]
		gpu_set_col(para->xs, n_features, Nmax, t, para->tmp_d_vec);

		// --- hs[t] = arma::tanh(Wxh * xs[t] + Whh*hs[t-1] + bh); ----
		// Wxh * xs[t]
		/*gpu_mv(handle, para->Wxh, para->tmp_d_vec, n_hidden, n_features, 
			para->W_tmp1); */// W_tmp1 saves Wxh*xs[t]
		// hs[t-1]
		gpu_get_col(para->hs, n_hidden, Nmax, t - 1 + 1, 
			para->tmp_d_vec2); // tmp2 saves hs[t-1]
		// Whh * hs[t-1]
		/*gpu_mv(handle, para->Whh, para->tmp_d_vec, n_hidden, n_hidden,
			para->W_tmp2);*/ // W_tmp2 saves Whh*hs[t-1]
		/*gpu_tanh_add_add(para->W_tmp1, para->W_tmp2, para->bh, n_hidden, 
			para->tmp_d_vec);*/ // tmp saves tanh_add_add
		gpu_tanh_Mv_add_Mv_add_v(handle,
			para->Wxh, n_hidden, n_features, para->tmp_d_vec,
			para->Whh, n_hidden, n_hidden, para->tmp_d_vec2, para->bh,
			para->W_tmp3, para);
		gpu_set_col(para->hs, n_hidden, Nmax, t + 1, para->W_tmp3);
		if (t == N-1)
		{
			// ys[t] = Why * hs[t] + by;
			gpu_get_col(para->hs, n_hidden, Nmax, t + 1, para->tmp_d_vec);
			gpu_mv(handle, para->Why, para->tmp_d_vec, n_output_classes, n_hidden, 
				para->W_tmp1); // Why * hs[t]
			gpu_add(para->W_tmp1, para->by, n_output_classes, para->tmp_d_vec);
			gpu_set_col(para->ys, n_output_classes, Nmax, t, para->tmp_d_vec);

			// ps[t] = softmax(ys[t])
			int sum1 = n_features + n_features + n_output_classes;
			gpu_clear_arr(para->W_tmp1, sum1*sum1);
			gpu_get_col(para->ys, n_output_classes, Nmax, t, para->tmp_d_vec);
			gpu_softmax(para->tmp_d_vec, n_output_classes, 
				para->W_tmp1, // dest
				para->W_tmp2); // cache
			gpu_set_col(para->ps, n_output_classes, Nmax, t, 
				para->W_tmp1); // W_tmp1 = softmax = ps[t]
			// loss += -log(ps[t](idx1));
			float val = para->W_tmp1[idx1_targets];
			loss += -logf(val);

			// idx_pred
			int idx_max_ps = gpu_max_index(para->W_tmp1, sum1*sum1, para->W_tmp2);

		}

	}

	// ---------------- BPTT -------------
	gpu_fill(para->dWxh, n_hidden*n_features, 0.f);
	gpu_fill(para->dWhh, n_hidden*n_hidden, 0.f);
	gpu_fill(para->dWhy, n_hidden*n_output_classes, 0.f);
	gpu_fill(para->dbh, n_hidden, 0.f);
	gpu_fill(para->dby, n_output_classes, 0.f);
	gpu_fill(para->dhnext, n_hidden, 0.f); // init dhnext = 0

	for (int t = N-1; t >= 0; t--)
	{
		if (t == N-1)
		{
			// dy = ps[t];
			gpu_get_col(para->ps, n_output_classes, Nmax, t, para->dy);
			// uvec fuvec = arma::find(targets == 1);
			// dy[fuvec(0)] -= 1;
			para->dy[idx1_targets] -= 1.f;
			// dWhy += dy * hs[t].t(); /// dy(10,1) * hs[t].t()(1,50) = (10,50)
			gpu_get_col(para->hs, n_hidden, Nmax, t + 1, 
				para->tmp_d_vec); // tmp saves hs[t]'
			gpu_mmul(handle, para->dy, para->tmp_d_vec, n_output_classes, 1, n_hidden, 
				para->W_tmp1); // Wtmp1 saves dy*hs[t]'
			gpu_add(para->dWhy, para->W_tmp1, n_output_classes*n_hidden, 
				para->dWhy);
			// dby += dy;
			gpu_add(para->dby, para->dy, n_output_classes, para->dby);


			// dh = Why.t() * dy + dhnext;
			gpu_mv(handle, para->Why, para->dy, n_output_classes, n_hidden,
				para->W_tmp1, true); // Wtmp1 saves Why' * dy
			gpu_add(para->W_tmp1, para->dhnext, n_hidden, para->dh);

			// dhraw = (1 - hs[t] % hs[t]) % dh; // mul elemwise
			gpu_get_col(para->hs, n_hidden, Nmax, t + 1, 
				para->tmp_d_vec); // tmp saves hs[t]
			gpu_tanh_der_hs_dh(para->tmp_d_vec, para->dh, n_hidden, 
				para->dhraw);

			// dbh += dhraw;
			gpu_add(para->dbh, para->dhraw, n_hidden, para->dbh);

			// dWxh += dhraw * xs[t].t(); // 惩罚项，只需要在loop中 加一次
			gpu_get_col(para->xs, n_features, Nmax, t, 
				para->tmp_d_vec); // tmp saves xs[t]
			gpu_mmul(handle, para->dhraw, para->tmp_d_vec, n_hidden, 1, n_features,
				para->W_tmp1); // Wtmp1 saves dhraw*xs[t]'
			gpu_add(para->dWxh, para->W_tmp1, n_hidden*n_features, 
				para->dWxh);

			// dWhh += dhraw * hs[t - 1].t();
			gpu_get_col(para->hs, n_hidden, Nmax, t-1+1,
				para->tmp_d_vec); // tmp saves hs[t-1]
			gpu_mmul(handle, para->dhraw, para->tmp_d_vec, n_hidden, 1, n_hidden,
				para->W_tmp1); // Wtmp1 saves dhraw*hs[t-1]'
			gpu_add(para->dWhh, para->W_tmp1, n_hidden*n_hidden,
				para->dWhh);

			// dhnext = Whh.t() * dhraw;
			gpu_mv(handle, para->Whh, para->dhraw, n_hidden, n_hidden, 
				para->dhnext, true);

		}
		else
		{
			// dh = dhnext;
			para->dh = para->dhnext;

			// dhraw = (1 - hs[t] % hs[t]) % dh; // mul elemwise
			gpu_get_col(para->hs, n_hidden, Nmax, t + 1,
				para->tmp_d_vec); // tmp saves hs[t]
			gpu_tanh_der_hs_dh(para->tmp_d_vec, para->dh, n_hidden,
				para->dhraw);

			// dbh += dhraw;
			gpu_add(para->dbh, para->dhraw, n_hidden, para->dbh);

			// dWxh += dhraw * xs[t].t(); // 惩罚项，只需要在loop中 加一次
			gpu_get_col(para->xs, n_features, Nmax, t,
				para->tmp_d_vec); // tmp saves xs[t]
			gpu_mmul(handle, para->dhraw, para->tmp_d_vec, n_hidden, 1, n_features,
				para->W_tmp1); // Wtmp1 saves dhraw*xs[t]'
			gpu_add(para->dWxh, para->W_tmp1, n_hidden*n_features,
				para->dWxh);

			// dWhh += dhraw * hs[t - 1].t();
			gpu_get_col(para->hs, n_hidden, Nmax, t - 1 + 1,
				para->tmp_d_vec); // tmp saves hs[t-1]
			gpu_mmul(handle, para->dhraw, para->tmp_d_vec, n_hidden, 1, n_hidden,
				para->W_tmp1); // Wtmp1 saves dhraw*hs[t-1]'
			gpu_add(para->dWhh, para->W_tmp1, n_hidden*n_hidden,
				para->dWhh);

			// dhnext = Whh.t() * dhraw;
			gpu_mv(handle, para->Whh, para->dhraw, n_hidden, n_hidden,
				para->dhnext, true);

		}

	}

	// clip
	gpu_clip(para->dWxh, n_hidden*n_features,-5.f, 5.f);
	gpu_clip(para->dWhh, n_hidden*n_hidden,-5.f, 5.f);
	gpu_clip(para->dWhy, n_hidden*n_output_classes,-5.f, 5.f);
	gpu_clip(para->dbh, n_hidden,-5.f, 5.f);
	gpu_clip(para->dby, n_output_classes,-5.f, 5.f);

}


float * score2onehot(float score, 
	int & idx1_targets, int n_output_classes, float score_min, float score_max)
{
	float part = 1.0f / n_output_classes;
	float pos = (score - score_min) / (score_max - score_min + pow(10, -8));

	idx1_targets = floor(pos / part);

	float* onehot;
	hipMallocManaged((void**)&onehot, n_output_classes * sizeof(float));
	// init onehot with 0
	gpu_fill(onehot, n_output_classes, 0.f);
	// set 1
	onehot[idx1_targets] = 1.f;

	return onehot;
}


void sgd(hipblasHandle_t handle, Para* para)
{
	int n_features = para->n_features[0];
	int n_hidden = para->n_hidden[0];
	int n_output_classes = para->n_output_classes[0];
	float alpha = para->alpha[0];

	sgd0(handle, para->Wxh, para->dWxh, n_hidden*n_features, alpha);
	sgd0(handle, para->Whh, para->dWhh, n_hidden*n_hidden, alpha);
	sgd0(handle, para->Why, para->dWhy, n_hidden*n_output_classes, alpha);
	sgd0(handle, para->bh,  para->dbh, n_hidden, alpha);
	sgd0(handle, para->by,  para->dby, n_output_classes, alpha);
}


void sgd0(hipblasHandle_t handle, float * P, float * dP,
	int size, float alpha)
{
	// P = - alpha * dP + P
	// hipblasSaxpy: y = a * x +  y

	//hipblasStatus_t stat;
	float a = -alpha;

	hipblasSaxpy(handle,
		size, // num of elems in P or dP
		&a,
		dP, 1,
		P, 1); // res into P
	//cout << "stat: " << stat << endl;

	hipDeviceSynchronize(); 
	// cublas执行后，必须跟一个同步，否则会因为数据同步问题报错。
}

void test_gpu_fns()
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// ===========================================
	const int M = 10;
	const int K = 11;
	const int N = 70;
	const int size1 = M*K;
	const int size2 = K*N;
	const int size3 = M*N;
	float* d_in1 = NULL;
	float* d_in2 = NULL;
	float* d_out = NULL;
	float* d_x;
	float* d_x2;
	float* d_x3;
	hipMallocManaged((void**)&d_in1, size1 * sizeof(float));
	hipMallocManaged((void**)&d_in2, size2 * sizeof(float));
	hipMallocManaged((void**)&d_out, size3 * sizeof(float));
	hipMallocManaged((void**)&d_x, K * sizeof(float));
	hipMallocManaged((void**)&d_x2, K * sizeof(float));
	hipMallocManaged((void**)&d_x3, K * sizeof(float));
	hipDeviceSynchronize();
	//printToHost(d_in1, M, K, "d_in1 initial");
	//printToHost(d_in2, K, N, "d_in2 initial");

	gpu_fill(d_in1, size1, 3.f);
	gpu_fill_rand(d_in2, size2, 1, -0.1f, 0.1f, 111);
	gpu_fill(d_x, K, 0.f); // d_x
	gpu_fill_rand(d_x2, 1, K, -4.f, 4.f, 43);
	gpu_fill_rand(d_x3, 1, K, 0.f, 1.f, 123);
	//printToHost(d_in1, M, K, "in1");
	//printToHost(d_in2, K, N, "rand2");
	d_x[1] = 0;
	//printToHost(d_x, 1, K, "x");
	printToHost(d_x2, 1, K, "x2");
	//printToHost(d_x3, 1, K, "x3");

	// --------- gpu_clip :)-------------
	/*gpu_clip(d_x2, K, -1.f, 1.f);
	printToHost(d_x2, 1, K, "clip x2");*/

	// ------ gpu_max_value -----------
	/*float* cache;
	hipMallocManaged((void**)&cache, K * sizeof(float));
	printToHost(cache, 1, K, "init cache");

	float x2_max = gpu_max_value(d_x2, K, cache);
	cout << "max val of x2: " << x2_max << endl;

	int idx = gpu_max_index(d_x2, K, cache);
	cout << "index of max val of x2: " << idx << endl;*/

	// ----------- gpu_sum -------------
	/*float* cache;
	hipMallocManaged((void**)&cache, K * sizeof(float));
	printToHost(cache, 1, K, "init cache");

	float s = gpu_sum(d_x, K, cache);
	cout << "sum of x: " << s << endl;

	printToHost(cache, 1, K+5, "cache");*/

	// --------- gpu_softmax -----------
	/*float* soft;
	hipMallocManaged((void**)&soft, K * sizeof(float));
	gpu_softmax(d_x, K, soft, cache);
	printToHost(soft, 1, K, "softmax of x");*/

	// ------------ gpu_scal -----------
	/*float* dest;
	hipMallocManaged((void**)&dest, K * sizeof(float));
	gpu_scal(d_x2, K, 0.1, dest);
	printToHost(dest, 1, K, "scal of x2");*/

	// -------- gpu_tanh_add_add :) --------------
	/*float* dest;
	hipMallocManaged((void**)&dest, K * sizeof(float));
	gpu_tanh_add_add(d_x, d_x2, d_x3, K, dest);
	printToHost(dest, 1, K, "tanh(v1+v2+v3)");*/
	
	// ------------ gpu_tanh :)------------------
	/*float* res_tanh;
	hipMallocManaged((void**)&res_tanh, K * sizeof(float));
	gpu_tanh(d_x, K, res_tanh);
	printToHost(res_tanh, 1, K, "tanh(x)");*/

	// ----------- gpu_mul_elemwise :)--------
	//float* mul;
	//hipMallocManaged((void**)&mul, M*K * sizeof(float));
	//gpu_mul_elemwise(d_in1, d_in1, M*K, mul);
	//printToHost(mul, M, K, "mul.");

	// ----------- gpu_add :) --------------------
	/*float* add;
	hipMallocManaged((void**)&add, M*K * sizeof(float));
	gpu_add(d_in1, d_in1, M*K, add);
	printToHost(add, M, K, "add");*/

	// -------------- gpu_mmul :)--------------
	/*gpu_mmul(handle, d_in1, d_in2, M, K, N, d_out);
	printToHost(d_out, M, N, "in1 * in2");*/
	
	// -------------- gpu_mv :)--------------
	/*float* Ax;
	hipMallocManaged((void**)&Ax, M * sizeof(float));
	gpu_mv(handle, d_in1, d_x, M, K, Ax, false);
	printToHost(Ax, M, 1, "Ax");*/

	// ------------ get get/set col :) -----------
	/*float* col1;
	hipMallocManaged((void**)&col1, M * sizeof(float));

	gpu_get_col(d_in, M, N, 1, col1);
	printToHost(col1, M, 1, "col1");

	float* setVal;
	hipMallocManaged((void**)&setVal, M, 1);
	gpu_fill(setVal, M * 1, 2.3f);
	gpu_set_col(d_in, M, N, 3, setVal);
	printToHost(d_in, M, N, "set col3 to 2.3");*/

	// --------- gpu_copy :) --------------
	/*float* d_cp = NULL;
	hipMallocManaged((void**)&d_cp, 2*M * sizeof(float));
	gpu_copy(d_cp, 5, d_in, 0, M);
	printToHost(d_cp, M, 2, "copy first col of d_in");*/

	// ----------- score2onehot :) -------
	/*int idx1;
	float* onehot = score2onehot(7.0f, idx1, 10, 6.0, 8.9);
	cout << "idx1: " << idx1 << endl;
	for (int i = 0; i < 10; i++)
	{
		cout << onehot[i] << "  " << endl;
	}
	cout << endl;*/

	// ------ gpu_clear_arr -----------
	//gpu_clear_arr(d_in, size);
	//printToHost(d_in, M, N, "clear to 0");

	// ------ fill rand :) --------

	/*gpu_fill_rand(d_in, size, 1, -0.1f, 0.1f, 11);
	printToHost(d_in, M, N, "rand");

	// ------- gpu_copy(); :) ---------------
	float* d_cp = NULL;
	hipMallocManaged((void**)&d_cp, 3 * sizeof(float));
	gpu_copy(d_cp, d_in, 1, 4);

	printToHost(d_cp, 1, 3, "copy");

	// ---------- gpu_fill(); :)-----------
	gpu_fill(d_in, size, 2.3);
	printToHost(d_in, M, N, "fill with 2.3");

	// free
	hipFree(d_cp);*/
	hipFree(d_in1);
	hipFree(d_in2);
	hipFree(d_out);
	hipblasDestroy(handle);
}

void initPara(Para * para, int Nmax)
{
	// malloc 
	hipHostMalloc((void**)&para->h_total_epoches, sizeof(float));
	hipHostMalloc((void**)&para->h_n_features, sizeof(float));
	hipHostMalloc((void**)&para->h_n_hidden, sizeof(float));
	hipHostMalloc((void**)&para->h_n_output_classes, sizeof(float));
	hipHostMalloc((void**)&para->h_alpha, sizeof(float));
	hipHostMalloc((void**)&para->h_score_min, sizeof(float));
	hipHostMalloc((void**)&para->h_score_max, sizeof(float));

	hipMalloc((void**)&para->d_total_epoches, sizeof(float));
	hipMalloc((void**)&para->d_n_features, sizeof(float));
	hipMalloc((void**)&para->d_n_hidden, sizeof(float));
	hipMalloc((void**)&para->d_n_output_classes, sizeof(float));
	hipMalloc((void**)&para->d_alpha, sizeof(float));
	hipMalloc((void**)&para->d_score_min, sizeof(float));
	hipMalloc((void**)&para->d_score_max, sizeof(float));

	// malloc dP, only in GPU needed
	hipMalloc((void**)&para->d_dWxh, n_hidden*n_features * sizeof(float));
	hipMalloc((void**)&para->d_dWhh, n_hidden*n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_dWhy, n_hidden*n_output_classes * sizeof(float));
	hipMalloc((void**)&para->d_dbh, n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_dby, n_output_classes * sizeof(float));
	hipMalloc((void**)&para->d_dhnext, n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_dy, n_output_classes * sizeof(float));
	hipMalloc((void**)&para->d_dh, n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_dhraw, n_hidden * sizeof(float));

	// malloc params of RNN
	hipHostMalloc((void**)&para->h_Wxh, n_hidden*n_features * sizeof(float));
	hipHostMalloc((void**)&para->h_Whh, n_hidden*n_hidden * sizeof(float));
	hipHostMalloc((void**)&para->h_Why, n_hidden*n_output_classes * sizeof(float));
	hipHostMalloc((void**)&para->h_bh, n_hidden * sizeof(float));
	hipHostMalloc((void**)&para->h_by, n_output_classes * sizeof(float));

	hipMalloc((void**)&para->d_Wxh, n_hidden*n_features * sizeof(float));
	hipMalloc((void**)&para->d_Whh, n_hidden*n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_Why, n_hidden*n_output_classes * sizeof(float));
	hipMalloc((void**)&para->d_bh, n_hidden * sizeof(float));
	hipMalloc((void**)&para->d_by, n_output_classes * sizeof(float));

	// malloc state, only in GPU needed
	hipMalloc((void**)&para->d_xs, n_features*Nmax * sizeof(float));
	hipMalloc((void**)&para->d_hs, n_hidden*(Nmax + 1) * sizeof(float));
	hipMalloc((void**)&para->d_ys, n_output_classes*Nmax * sizeof(float));
	hipMalloc((void**)&para->d_ps, n_output_classes*Nmax * sizeof(float));
	hipMallocManaged((void**)&para->d_Nmax, sizeof(float));
	para->d_Nmax[0] = Nmax;
	
	// malloc cache
	int sum1 = n_features + n_features + n_output_classes;
	hipMalloc((void**)&para->d_tmp_d_vec, sum1 * sizeof(float));
	hipMalloc((void**)&para->d_tmp_d_vec2, sum1 * sizeof(float));
	hipMalloc((void**)&para->d_W_tmp1, sum1*sum1 * sizeof(float));
	hipMalloc((void**)&para->d_W_tmp2, sum1*sum1 * sizeof(float));
	hipMalloc((void**)&para->d_W_tmp3, sum1*sum1 * sizeof(float));
	

}

void deInitPara(Para * para)
{
	// free 
	hipHostFree(para->h_total_epoches);
	hipHostFree(para->h_n_features);
	hipHostFree(para->h_n_hidden);
	hipHostFree(para->h_n_output_classes);
	hipHostFree(para->h_alpha);
	hipHostFree(para->h_score_min);
	hipHostFree(para->h_score_max);

	hipFree(para->d_total_epoches);
	hipFree(para->d_n_features);
	hipFree(para->d_n_hidden);
	hipFree(para->d_n_output_classes);
	hipFree(para->d_alpha);
	hipFree(para->d_score_min);
	hipFree(para->d_score_max);

	// free dP, only GPU
	hipFree(para->d_dWxh);
	hipFree(para->d_dWhh);
	hipFree(para->d_dWhy);
	hipFree(para->d_dbh);
	hipFree(para->d_dby);
	hipFree(para->d_dhnext);
	hipFree(para->d_dy);
	hipFree(para->d_dh);
	hipFree(para->d_dhraw);

	// free params of RNN
	hipHostFree(para->h_Wxh);
	hipHostFree(para->h_Whh);
	hipHostFree(para->h_Why);
	hipHostFree(para->h_bh);
	hipHostFree(para->h_by);

	hipFree(para->d_Wxh);
	hipFree(para->d_Whh);
	hipFree(para->d_Why);
	hipFree(para->d_bh);
	hipFree(para->d_by);

	// free sces
	hipHostFree(para->h_sces_id_score);
	hipHostFree(para->h_sces_data);
	hipHostFree(para->h_sces_data_mn);
	hipHostFree(para->h_sces_data_idx_begin);
	hipHostFree(para->h_num_sces);
	hipHostFree(para->h_total_epoches);

	hipFree(para->d_sces_id_score);
	hipFree(para->d_sces_data);
	hipFree(para->d_sces_data_mn);
	hipFree(para->d_sces_data_idx_begin);
	hipFree(para->d_num_sces);
	hipFree(para->d_total_epoches);

	// free state, inly GPU
	hipFree(para->d_xs);
	hipFree(para->d_hs);
	hipFree(para->d_ys);
	hipFree(para->d_ps);
	hipFree(para->d_Nmax);

	// free cache, only  GPU
	int sum1 = n_features + n_features + n_output_classes;
	hipFree(para->d_tmp_d_vec);
	hipFree(para->d_tmp_d_vec2);
	hipFree(para->d_W_tmp1);
	hipFree(para->d_W_tmp2);
	hipFree(para->d_W_tmp3);
}

