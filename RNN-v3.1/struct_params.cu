#include "hip/hip_runtime.h"
﻿#include "struct_params.h"

__global__ void add_kernel(Para* para, int size)
{
	int tid = threadIdx.x;

	if (tid < size)
	{
		para->d_c[tid] = para->d_a[tid] + para->d_b[tid];
	}
}

void struct_para_main()
{
	const int size = 10;
	const int bytes = size * sizeof(float);

	// alloc struct on the host
	Para* h_para; // struct 只需要在host端定义即可
	hipHostMalloc((void**)&h_para, sizeof(Para));

	// alloc params in struct on the host/dev
	hipHostMalloc((void**)&h_para->h_a, bytes);
	hipHostMalloc((void**)&h_para->h_b, bytes);
	hipHostMalloc((void**)&h_para->h_c, bytes);
	hipMalloc((void**)&h_para->d_a, bytes);
	hipMalloc((void**)&h_para->d_b, bytes);
	hipMalloc((void**)&h_para->d_c, bytes);

	// init values of h_a, h_b
	for (int i = 0; i < size; i++)
	{
		h_para->h_a[i] = 1.f;
		h_para->h_b[i] = 2.f;
	}

	// mem cpy host 2 dev
	hipMemcpy(h_para->d_a, h_para->h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(h_para->d_b, h_para->h_b, bytes, hipMemcpyHostToDevice);

	add_kernel << <1, size >> > (h_para, size);

	// mem cpy dev 2 host
	hipMemcpy(h_para->h_c, h_para->d_c, bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < size; i++)
	{
		cout << h_para->h_c[i] << "  ";
	}
	cout << endl;




	// free
	hipFree(h_para->d_a);
	hipFree(h_para->d_b);
	hipFree(h_para->d_c);
	hipHostFree(h_para->h_a);
	hipHostFree(h_para->h_b);
	hipHostFree(h_para->h_c);
	hipHostFree(h_para);
}
